
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <inttypes.h>
#include <string.h>
#include <time.h>
#include <math.h>

//Cuda error checking - non mandatory
void cudaCheckError() {
 hipError_t e=hipGetLastError();
 if(e!=hipSuccess) {
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e));
   exit(0); 
 }
}


int16_t op_x[9]={
    -1, 0, 1,
    -2, 0, 2,
    -1 ,0, 1  
}; 

int16_t op_y[9]={
    -1,-2,-1,
     0, 0, 0,
     1, 2, 1  
}; 

__global__
void filtro_pixel(int16_t* matriz_entrada, int16_t* opx, int16_t* opy, int16_t* matriz_salida){
    int row = blockIdx.x;
    int col = threadIdx.x;
    int cols = blockDim.x;
    int rows = gridDim.x;

    // curr = row*col - 1
    // prev = (row-1)*col - 1
    // post = (row+1)*col - 1
    int curr = row*cols + col-1;
    if(col == 0 || col == (cols-1) || row ==0 || row ==(rows-1)){
        // no procesar bordes
        *(matriz_salida + curr+1) = *(matriz_entrada + curr+1);
        return;
    }
    int16_t* prow0 = matriz_entrada + curr - cols;
    int16_t* prow1 = matriz_entrada + curr;
    int16_t* prow2 = matriz_entrada + curr + cols;
    int16_t result;
    int resultx = 0;
    int resulty = 0; 
    //opx y opy tienen que se copiadas a mem de la gpu duuuhh

    for (int i=0; i<3; i++){
        resultx += *(prow0+i) * opx[0+i];
        resultx += *(prow1+i) * opx[3+i];
        resultx += *(prow2+i) * opx[6+i];
        resulty += *(prow0+i) * opy[0+i];
        resulty += *(prow1+i) * opy[3+i];
        resulty += *(prow2+i) * opy[6+i];
    }
    result = (int16_t)(sqrt((double)(resultx*resultx + resulty*resulty)));
    if(result > 255) result = 255;
    matriz_salida[curr+1] = (result);
}

// Recibe la imagen con padding, y la guarda con padding 
void filtro(int16_t* matriz_entrada, int rows, int cols, int16_t* matriz_salida){

    int16_t* mat_entrada_gpu;
    int16_t* mat_salida_gpu;
    int16_t* opx;
    int16_t* opy;
    int pixel_count=rows*cols;

    int mem_size = pixel_count * sizeof(int16_t);

    // reservar memoria en gpu
    hipMalloc((void **) &mat_entrada_gpu, mem_size);
    hipMalloc((void **) &mat_salida_gpu, mem_size);
    hipMalloc((void **) &opx, 9*sizeof(int16_t));
    hipMalloc((void **) &opy, 9*sizeof(int16_t));

    // copiar datos de entrada para kernel a memoria de gpu
    hipMemcpy(mat_entrada_gpu, matriz_entrada, mem_size, hipMemcpyHostToDevice);
    hipMemcpy(opx,op_x, sizeof(op_x), hipMemcpyHostToDevice);
    hipMemcpy(opy,op_y, sizeof(op_y), hipMemcpyHostToDevice);
    
    dim3 dimBlock(rows,1,1);
    dim3 dimGrid(cols,1,1);

    // ejercutar kernel
    filtro_pixel<<<rows,cols>>>(mat_entrada_gpu, opx, opy, mat_salida_gpu);
    hipDeviceSynchronize();
    cudaCheckError();

    // copiar resultados a la matriz de salida
    hipMemcpy(matriz_salida, mat_salida_gpu, mem_size, hipMemcpyDeviceToHost);

    // liberar recursos 
    hipFree(mat_entrada_gpu);
    hipFree(mat_salida_gpu);
    hipFree(opx);
    hipFree(opy);

}

int main(int argc, char** argv){
    if(argc < 3){
        fprintf(stderr, "No se tienen suficientes parámetros");
        return 1;
    }
    FILE *input = fopen(argv[1], "r");
    if(input == NULL){
        return 2;
    }
    FILE *output = fopen(argv[2], "w"); 
    if(output == NULL){
        return 2;
    }
    int rows;
    int cols;
    if(fscanf(input, "%d\n", &rows)==EOF) return 3;
    if(fscanf(input, "%d\n", &cols) ==EOF) return 3;
    int pixel_count = rows*cols; 
    
    int16_t* src_image = (int16_t*)calloc((pixel_count), sizeof(int16_t));
    int16_t* out_image = (int16_t*)calloc(pixel_count, sizeof(int16_t));

    if(src_image == NULL || out_image == NULL) return 3;
    int i = 0;
    int pixel;
    
    while(fscanf(input, "%d\n", &pixel)!=EOF){
        src_image[i] = pixel;
        ++i;
    }
 
    // Los movimientos de memoria son parte del tradeoff. La carga del archivo a memoria de CPU no
    // cuenta, pero para ser justos, el movimiento de datos a memoria de GPU si se va a contar
    clock_t start = clock();
    filtro(src_image, rows, cols, out_image);
    clock_t end = clock();
    printf("Filtro tomó: %lf segundos\n", ((double)(end - start))/CLOCKS_PER_SEC);
    

    fprintf(output, "%d\n%d\n", rows, cols);
    for(int i=0; i < pixel_count; i++){
        fprintf(output, "%d\n", *(out_image+i));
    }
    free(src_image);
    free(out_image);
}
